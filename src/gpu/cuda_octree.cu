#include "hip/hip_runtime.h"
//
// Created by Mike on 6/16/2021.
//

#include <glm/gtx/component_wise.hpp>
#include <gpu/cuda_octree.h>

[[nodiscard]] SV_XPU inline auto _intersect_box(Ray ray, glm::vec3 bbox_min, float bbox_r) noexcept {
    auto bbox_max = bbox_min + bbox_r;
    auto t_min = (bbox_min - ray.o) / ray.d;
    auto t_max = (bbox_max - ray.o) / ray.d;
    auto o_mat = glm::mat3{ray.o, ray.o, ray.o};
    glm::vec3 p_min[]{ray.o + t_min.x * ray.d, ray.o + t_min.y * ray.d, ray.o + t_min.z * ray.d};
    glm::vec3 p_max[]{ray.o + t_max.x * ray.d, ray.o + t_max.y * ray.d, ray.o + t_max.z * ray.d};
    auto valid = [ray, bbox_min, bbox_max](auto t, auto p) noexcept {
        return glm::not_(glm::isnan(t))
               && glm::greaterThanEqual(t, glm::vec3{ray.t_min})
               && glm::lessThanEqual(t, glm::vec3{ray.t_max})
               && glm::bvec3{p[0].y >= bbox_min.y && p[0].y <= bbox_max.y
                                 && p[0].z >= bbox_min.z && p[0].z <= bbox_max.z,
                             p[1].z >= bbox_min.z && p[1].z <= bbox_max.z
                                 && p[1].x >= bbox_min.x && p[1].x <= bbox_max.x,
                             p[2].x >= bbox_min.x && p[2].x <= bbox_max.x
                                 && p[2].y >= bbox_min.y && p[2].y <= bbox_max.y};
    };
    auto t_invalid = glm::vec3{std::numeric_limits<float>::max()};
    auto valid_min = valid(t_min, p_min);
    auto valid_max = valid(t_max, p_max);
    t_min = glm::mix(t_invalid, t_min, valid_min);
    t_max = glm::mix(t_invalid, t_max, valid_max);
    auto t = glm::min(t_min, t_max);
    auto is_min = glm::lessThanEqual(t_min, t_max);
    Hit hit{};
    hit.t = std::numeric_limits<float>::max();
    hit.valid = false;
    if (t.x < hit.t) {
        hit.p = is_min.x ? p_min[0] : p_max[0];
        hit.t = t.x;
        hit.ng = {is_min.x ? -1.0f : 1.0f, 0.0f, 0.0f};
        hit.valid = valid_min.x || valid_max.x;
    }
    if (t.y < hit.t) {
        hit.p = is_min.y ? p_min[1] : p_max[1];
        hit.t = t.y;
        hit.ng = {0.0f, is_min.y ? -1.0f : 1.0f, 0.0f};
        hit.valid = valid_min.y || valid_max.y;
    }
    if (t.z < hit.t) {
        hit.p = is_min.z ? p_min[2] : p_max[2];
        hit.t = t.z;
        hit.ng = {0.0f, 0.0f, is_min.z ? -1.0f : 1.0f};
        hit.valid = valid_min.z || valid_max.z;
    }
    return hit;
}

static constexpr auto m000 = 0b00000001u;
static constexpr auto m001 = 0b00000010u;
static constexpr auto m011 = 0b00000100u;
static constexpr auto m010 = 0b00001000u;
static constexpr auto m110 = 0b00010000u;
static constexpr auto m111 = 0b00100000u;
static constexpr auto m101 = 0b01000000u;
static constexpr auto m100 = 0b10000000u;
__constant__ uint32_t _node_m[]{m000, m001, m011, m010, m110, m111, m101, m100};

static constexpr auto d000 = float3{0.0f, 0.0f, 0.0f};
static constexpr auto d001 = float3{0.0f, 0.0f, 1.0f};
static constexpr auto d010 = float3{0.0f, 1.0f, 0.0f};
static constexpr auto d011 = float3{0.0f, 1.0f, 1.0f};
static constexpr auto d100 = float3{1.0f, 0.0f, 0.0f};
static constexpr auto d101 = float3{1.0f, 0.0f, 1.0f};
static constexpr auto d110 = float3{1.0f, 1.0f, 0.0f};
static constexpr auto d111 = float3{1.0f, 1.0f, 1.0f};
__constant__ float3 _node_d[]{d000, d001, d011, d010, d110, d111, d101, d100};

__global__ void octree_trace_closest(const CUDAOctree::Node *nodes, uint32_t resolution, const Ray *rays, Hit *hits, uint32_t w, uint32_t h) {

    auto x = threadIdx.x + blockIdx.x * blockDim.x;
    auto y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x >= w || y >= h) { return; }

    auto tid = x + y * w;

    Hit closest{};
    closest.t = std::numeric_limits<float>::max();
    closest.valid = false;
    if (nodes[0].empty()) {
        hits[tid] = closest;
        return;
    }
    
    struct alignas(16) TraceContext {
        glm::vec3 o;
        float r;
        uint32_t index;
        float t;
    };

    static constexpr auto stack_size = 32u;
    TraceContext stack[stack_size];
    auto sp = 0u;

    auto ray = rays[tid];

    auto add_node = [&sp, ray, &stack, &closest, nodes](auto index, auto o, auto r) noexcept {
        if (auto hit = _intersect_box(ray, o, r); hit.valid && hit.t < closest.t) {
            auto node = nodes[index];
            if (node.full()) {
                closest = hit;
            } else if (r == 2.0f) {
                #pragma unroll
                for (auto i = 0u; i < 8u; i++) {
                    if ((node.child_masks() & _node_m[i])) {
                        auto d = _node_d[i];
                        if (auto child_hit = _intersect_box(ray, o + glm::vec3{d.x, d.y, d.z}, 1.0f);
                            child_hit.valid && child_hit.t < closest.t) {
                            closest = child_hit;
                        }
                    }
                }
            } else {
                if (sp == stack_size) { printf("warning: stack overflows\n"); }
                stack[sp++] = {o, r, index, hit.t};
            }
        }
    };

    add_node(0u, glm::vec3{}, static_cast<float>(resolution));
    while (sp != 0u) {
        auto ctx = stack[--sp];
        if (ctx.t >= closest.t) { continue; }
        auto node = nodes[ctx.index];
        auto half_r = ctx.r * 0.5f;
        #pragma unroll
        for (auto i = 0u; i < 8u; i++) {
            if (node.child_masks() & _node_m[i]) {
                auto d = _node_d[i];
                add_node(ctx.index + node.child_offset() + i, ctx.o + glm::vec3{d.x, d.y, d.z} * half_r, half_r);
            }
        }
    }

    hits[tid] = closest;
}

void CUDAOctree::trace_closest(const Ray *rays, Hit *hits, uint32_t width, uint32_t height) const noexcept {
    static constexpr auto block_size = 16u;
    auto blocks_x = (width + block_size - 1u) / block_size;
    auto blocks_y = (height + block_size - 1u) / block_size;
    octree_trace_closest<<<dim3(blocks_x, blocks_y), dim3(block_size, block_size)>>>(_nodes->data(), _resolution, rays, hits, width, height);
}
