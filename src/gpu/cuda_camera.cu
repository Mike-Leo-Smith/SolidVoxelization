#include "hip/hip_runtime.h"
#include <gpu/cuda_camera.h>

template<uint32_t times>
[[nodiscard]] SV_XPU auto tea(glm::uvec2 xy) noexcept {
    auto v0 = xy.x;
    auto v1 = xy.y;
    auto s0 = 0u;
    for (auto n = 0u; n < times; n++) {
        s0 += 0x9e3779b9u;
        v0 += ((v1 << 4u) + 0xa341316cu) ^ (v1 + s0) ^ ((v1 >> 5u) + 0xc8013ea4u);
        v1 += ((v0 << 4u) + 0xad90777du) ^ (v0 + s0) ^ ((v0 >> 5u) + 0x7e95761eu);
    }
    return v0;
};

template<uint32_t base>
[[nodiscard]] SV_XPU auto halton(uint32_t i) noexcept {
    auto f = 1.0f;
    auto inv_base = 1.0f / base;
    auto r = 0.0f;
    while (i > 0u) {
        f = f * inv_base;
        r = r + f * (i % base);
        i /= base;
    }
    return r;
}

[[nodiscard]] SV_XPU auto pixel_sample(glm::uvec2 p, uint32_t frame) noexcept {
    auto index = tea<4u>(p) + frame;
    auto x = halton<2>(index);
    auto y = halton<3>(index);
    return glm::vec2{x, y};
}

__global__ void generate_rays(Ray *rays, uint32_t w, uint32_t h, uint32_t frame, glm::vec3 o, glm::mat3 m, float pixel_scale, float z_plane) {
    auto x = threadIdx.x + blockIdx.x * blockDim.x;
    auto y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x >= w || y >= h) { return; }
    auto tid = y * w + x;
    auto xy = glm::uvec2{x, y};
    auto p = pixel_sample(xy, frame) + glm::vec2{xy};
    auto d = glm::normalize(m * glm::vec3{p.x * pixel_scale - 1.0f, 1.0f - p.y * pixel_scale, -z_plane});
    rays[tid] = {o, 0.0f, d, std::numeric_limits<float>::max()};
}

void cuda_camera_generate_rays(
    Camera cam, glm::mat4 world_to_object,
    Ray *rays, uint32_t w, uint32_t h, uint32_t frame) noexcept {
    auto o = glm::vec3{world_to_object * glm::vec4{cam.position(), 1.0f}};
    auto m = glm::mat3{world_to_object} * cam.rotation_to_world();
    auto pixel_scale = cam.pixel_scale();
    auto z_plane = cam.z_plane();
    static constexpr auto block_size = 16u;
    auto bx = (w + block_size - 1u) / block_size;
    auto by = (h + block_size - 1u) / block_size;
    generate_rays<<<dim3(bx, by), dim3(block_size, block_size)>>>(rays, w, h, frame, o, m, pixel_scale, z_plane);
}